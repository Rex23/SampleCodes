#include "hip/hip_runtime.h"
#include "Event_Create_Record_Synchonize.cuh"

__global__ void saxpy_kernel(const int N, const float a, float* d_x, float* d_y)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < N) {
		d_y[i] = a * d_x[i] + d_y[i];
	}
}

int Event_Create_Record_Synchonize(void)
{
	int N = 10000;

	float *x, *y, *d_x, *d_y;

	x = (float *)malloc(sizeof(float) * N);
	y = (float *)malloc(sizeof(float) * N);

	for (auto m = 0; m < N; m++) {
		x[m] = 1.0; y[m] = 1.0;
	}

	hipMalloc(&d_x, sizeof(float)*N);
	hipMalloc(&d_y, sizeof(float)*N);

	hipMemcpy(d_x, x, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, sizeof(float) * N, hipMemcpyHostToDevice);

	const int ThreadsPerBlock = 256;
	dim3 grid((N + ThreadsPerBlock - 1) / ThreadsPerBlock);
	dim3 block(ThreadsPerBlock);

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	saxpy_kernel <<< grid, block >>> (N, 2.0, d_x, d_y);

	hipEventRecord(stop);

	hipMemcpy(y, d_y, sizeof(float) * N, hipMemcpyDeviceToHost);

	//for (auto m = 0; m < 10000; m++)
	//	cout << "Test y: " << y[m] << endl;

	hipEventSynchronize(stop);

	float milliseconds = 0.0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << "Time Elapsed: " << milliseconds << endl;

	free(x);
	free(y);
	hipFree(d_x);
	hipFree(d_y);

	return 1;
}