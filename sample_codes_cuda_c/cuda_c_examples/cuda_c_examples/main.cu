#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include "devicequery.cuh"
#include "kernel.cuh"
#include "dot_product.cuh"
#include "const_memory.cuh"
#include "Event_Create_Record_Synchonize.cuh"
#include "non_default_streams.cuh"
#include "cublas_one_based_indexing.cuh"

using namespace std;

int main(int argc, char** argv)
{
	int a_case = 6;

	//Sample code: Query Device Properties
	devicequery(argc, argv);

	switch (a_case) {
	case 1:
		//Sample code: Add two arrays
		add_two_arrays_example();
		break;
	case 2:
		//Sample code: Dot product of two vectors. The shared memory is used for each block.
		dot_product();
		break;
	case 3:
		//Sample code: Constant memory usage
		const_memory();
		break;
	case 4:
		//Sample code: Cuda events creat, record and synchonize
		Event_Create_Record_Synchonize();
		break;
	case 5:
		//Sample code: Cuda non-default streams
		non_default_streams();
		break;
	case 6:
		//Sample code: Cublas for one based indexing
		cblas_one_based_indexing();
	default:
		break;
	}

	system("pause");

	// finish
	exit(EXIT_SUCCESS);

	return 1;
}

