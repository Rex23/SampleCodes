#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include "devicequery.cuh"
#include "kernel.cuh"
#include "dot_product.cuh"
#include "const_memory.cuh"

using namespace std;

int main(int argc, char** argv)
{
	//Sample code 1: Query Device Properties
	devicequery(argc, argv);
	
	//Sample code 2: Add two arrays
	//add_two_arrays_example();
	
	//Sample code 3: Dot product of two vectors. The shared memory is used for each block.
	//dot_product();

	//Sample code 4: Constant memory usage
	const_memory();

	system("pause");

	// finish
	exit(EXIT_SUCCESS);

	return 1;
}

