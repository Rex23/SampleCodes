#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include "devicequery.cuh"
#include "kernel.cuh"
#include "dot_product.cuh"

using namespace std;

int main(int argc, char** argv)
{
	//Sample code 1: Query Device Properties
	//devicequery(argc, argv);
	
	//Sample code 2: Add two arrays
	//add_two_arrays_example();
	
	//Sample code 3: Dot product of two vectors. The shared memory is used for each block.
	dot_product();

	system("pause");

	// finish
	exit(EXIT_SUCCESS);

	return 1;
}

